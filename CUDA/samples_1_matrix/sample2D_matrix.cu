
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void init7(int* arr,size_t pitch,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	int idy=blockDim.y*blockIdx.y+threadIdx.y;
	if(idx <n && idy<n){
		int* row=(int*)( (char*)arr +idy*pitch);
		row[idx]=7;
	}
}

__host__ void printMtrx(int* const mtrx,const int n){
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++)
			printf("%d ",mtrx[i*n+j]);
		printf("\n");
	}
	printf("\n");
}

__host__ int main(int argc,char* argv[]){
	const int n = 5;
	size_t pitch;
	size_t size=n*sizeof(n);
	int* devPtr=NULL;
	hipMallocPitch((void**)&devPtr,&pitch,size,n);
	
	int* mt1=(int*)malloc(n*size);
	memset(mt1,0,n*size);
	
	dim3 grid(n,n,1);
	init7<<<grid,1>>>(devPtr,pitch,n);
	hipMemcpy2D(mt1,size,devPtr,pitch,size,n,hipMemcpyDeviceToHost);
	printMtrx(mt1,n);
	hipFree(devPtr);
	free(mt1);
	return 0;
}
