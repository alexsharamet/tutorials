
#include <hip/hip_runtime.h>
#include <cstdio>
#define BLOCK_SIZE 256

__global__ void mtrxToVec(int* mtrx,int* vec, int* ans,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	while(idx<n){
		int sum=0;
		for(int i=0;i<n;i++)
			sum+=mtrx[n*i+idx]*vec[idx];
		ans[idx]=sum;
		idx+=gridDim.x*blockDim.x;
	}
	return;
}


__global__ void mtrxToVecY(int* mtrx,int* vec, int* ans,int n){
	int idy=blockDim.y*blockIdx.y+threadIdx.y;
	while(idy<n){
		int sum=0;
		for(int i=0;i<n;i++)
			sum+=mtrx[n*i+idy]*vec[idy];
		ans[idy]=sum;
		idy+=gridDim.y*blockDim.y;
	}
	return;
}

__global__ void transposeMtrxToVec(int* mtrx,int* vec, int* ans,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	while(idx<n){
		int sum=0;
		for(int i=0;i<n;i++)
			sum+=mtrx[n+idx*i]*vec[idx];
		ans[idx]=sum;
		idx+=gridDim.x*blockDim.x;
	}
	return;
}

__host__ void  cuAssert(hipError_t error){
	if(error!=hipSuccess){
		printf("%s\n",hipGetErrorString(error));
		exit(0);
	}
}

__host__ void print(int* ans,int n){
	for(int i=0;i<n;i++)
		printf("%5d ",ans[i]);
	printf("\n");
	return;
}

__host__ int main(int argc,char* argv[]){
	hipEvent_t start,stop;
	cuAssert(hipEventCreate(&start));
	cuAssert(hipEventCreate(&stop));
	float time;

	const int n=10000;
	size_t size=sizeof(int)*n;
	int* vec=(int*)malloc(size);
	int* ans=(int*)malloc(size);
	int* mtrx=(int*)malloc(size*n);

	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++)
			mtrx[i*n+j]=(i==j)?(1):(0);
		vec[i]=i;
	}

	int* cuVec=NULL;
	int* cuAns=NULL;
	int* cuMtrx=NULL;
	cuAssert(hipMalloc((void**)&cuVec,size));
	cuAssert(hipMalloc((void**)&cuAns,size));
	cuAssert(hipMalloc((void**)&cuMtrx,size*n));

	cuAssert(hipMemcpy(cuMtrx,mtrx,size*n,hipMemcpyHostToDevice));
	cuAssert(hipMemcpy(cuVec,vec,size,hipMemcpyHostToDevice));

	dim3 grid((n+BLOCK_SIZE-1)/BLOCK_SIZE,1,1);
	dim3 block(BLOCK_SIZE,1,1);

//coalesced block 1D indexes
	cuAssert(hipEventRecord(start));
	mtrxToVec<<<grid,block>>>(cuMtrx,cuVec,cuAns,n);
	cuAssert(hipEventRecord(stop));
	cuAssert(hipEventSynchronize(stop));
	cuAssert(hipEventElapsedTime(&time,start,stop));
	printf("coalesced time X %f\n",time);
	cuAssert(hipGetLastError());
	cuAssert(hipMemcpy(ans,cuAns,size,hipMemcpyDeviceToHost));
	//print(ans,n);

	dim3 gridY(1,(n+BLOCK_SIZE-1)/BLOCK_SIZE,1);
	dim3 blockY(1,BLOCK_SIZE,1);	
	cuAssert(hipEventRecord(start));
	mtrxToVecY<<<gridY,blockY>>>(cuMtrx,cuVec,cuAns,n);
	cuAssert(hipEventRecord(stop));
	cuAssert(hipEventSynchronize(stop));
	cuAssert(hipEventElapsedTime(&time,start,stop));
	printf("coalesced time Y %f\n",time);
	cuAssert(hipGetLastError());
	
//Non coalesced block
	cuAssert(hipEventRecord(start));
	transposeMtrxToVec<<<grid,block>>>(cuMtrx,cuVec,cuAns,n);
	cuAssert(hipEventRecord(stop));
	cuAssert(hipEventSynchronize(stop));
	cuAssert(hipEventElapsedTime(&time,start,stop));
	printf("non coalesced time %f\n",time);
	cuAssert(hipGetLastError());

	cuAssert(hipFree(cuVec));
	cuAssert(hipFree(cuAns));
	cuAssert(hipFree(cuMtrx));

	free(vec);
	free(ans);
	free(mtrx);
	return 0;
}
