
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void init5(int* arr, int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	if(idx<n*n)
		arr[idx]=5;
}

__global__ void init7(int* arr,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	int idy=blockDim.y*blockIdx.y+threadIdx.y;

	arr[idy*n+idx]=7;
}

__host__ void printMtrx(int* const mtrx,const int n){
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++)
			printf("%d ",mtrx[i*n+j]);
		printf("\n");
	}
	printf("\n");
}

__host__ int main(int argc,char* argv[]){
	const int n = 5;
	size_t bytes=sizeof(int)*n*n;

	int* cuMtrx=NULL;
	hipMalloc((void**)&cuMtrx,bytes);

	int* mt1=(int*)malloc(bytes);
	memset(mt1,0,bytes);

	hipMemcpy(cuMtrx,mt1,bytes,hipMemcpyHostToDevice);
	init5<<<n*n,1>>>(cuMtrx,n);
	hipMemcpy(mt1,cuMtrx,bytes,hipMemcpyDeviceToHost);
	printMtrx(mt1,n);

	dim3 grid(n,n,1);
	init7<<<grid,1>>>(cuMtrx,n);
	hipMemcpy(mt1,cuMtrx,bytes,hipMemcpyDeviceToHost);
	printMtrx(mt1,n);

	hipFree(cuMtrx);

	return 0;
}
