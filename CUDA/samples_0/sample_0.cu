
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>

__global__ void kernel(int* arr,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	if(idx<n){
		arr[idx]=5;
	}
	return;
}

__host__ int main(int argc,char* argv[]){
	int* arr=NULL;
	int* cuArr=NULL;
	const int n=100;
	size_t size=n*sizeof(int);
	
	arr=(int*)malloc(size);
	hipMalloc((void**)&cuArr,size);

	kernel<<<2,64>>>(cuArr,n);
	hipMemcpy(arr,cuArr,size,hipMemcpyDeviceToHost);
	hipFree(cuArr);
	
	for(int i=0;i<n;i++){
		printf("%d ",arr[i]);
	}
	printf("\n");
	free(arr);

	return 0;
}
