
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <string>
#include <iostream>

__global__ void kernel(int* arr,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	if(idx<n){
		arr[idx]=5;
	}
	return;
}

__host__ void  error(std::string message,bool warning=false){
	hipError_t err=hipSuccess;
	err=hipGetLastError();
	if(err!=hipSuccess){
		std::cerr<<message<<std::endl;
		if(warning)
			return;
		exit(0);
	}

	return;
}

__host__ int main(int argc,char* argv[]){
	int* arr=NULL;
	int* cuArr=NULL;
	const int n=100;
	size_t size=n*sizeof(int);
	
	arr=(int*)malloc(size);
	hipMalloc((void**)&cuArr,size);

	kernel<<<2,64>>>(cuArr,n);
	hipMemcpy(arr,cuArr,size,hipMemcpyDeviceToHost);
	hipFree(cuArr);
	
	for(int i=0;i<n;i++){
		std::cout<<arr[i]<<" ";
	}
	std::cout<<std::endl;
	free(arr);

	return 0;
}
