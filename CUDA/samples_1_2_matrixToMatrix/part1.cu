
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstdlib>
#include <cstdio>

__host__ void  matrixMult(int* const a,int* const b,int* const c,const int n){
	for(int i=0;i<n;i++)
		for(int j=0;j<n;j++)
			for(int k=0;k<n;k++)
				c[i*n+j]+=a[i*n+k]*b[k*n+j];
	return;
}

__host__ void printMatrix(const int* const a,const int n){
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++)
			printf("%d ",a[i*n+j]);
		printf("\n");
	}
	printf("\n");
}

__host__ void init(int* const a,const int n){
	for(int i=0;i<n;i++)
		for(int j=0;j<n;j++)
			a[i*n+j]=rand()%10;	
	return;
}

__host__ bool validate(const int* const a,const int* const  b,const int n){
	for(int i=0;i<n;i++)
		for(int j=0;j<n;j++)
			if(a[i*n+j]!=b[i*n+j])
				return false;
	return true;
}

 __host__ void  cuAssert(hipError_t error){
     if(error!=hipSuccess){
         printf("%s\n",hipGetErrorString(error));
         exit(0);
     }
 }

__host__ void printTime(char* const s,struct timespec tstart,struct timespec tend){
    printf("%s %.5f seconds\n",s,
           ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
           ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec));
}  

__global__ void memsetGPUKernel(int* arr,int n){
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    int idy=blockDim.y*blockIdx.y+threadIdx.y;
    if(idx<n && idy<n)
        arr[idy*n+idx]=0;
    return;
}

__host__ void memsetGPU(int* arr,int n){
    int k=(n+15)/16;
    dim3 grid(k,k);
    dim3 block(16,16);
    memsetGPUKernel<<<grid,block>>>(arr,n);
    cuAssert(hipGetLastError());
}

__global__ void naive(int* a,int* b,int* c,int n){
	int	idx=blockDim.x*blockIdx.x+threadIdx.x;
	int idy=blockDim.y*blockIdx.y+threadIdx.y;
	if(idx<n && idy<n){
		for(int k=0;k<n;k++)
			c[idx*n+idy]+=a[idx*n+k]*b[k*n+idy];
	}
}

__global__ void common(int* a,int* b,int* c,int n){
	int	idx=blockDim.x*blockIdx.x+threadIdx.x;
	int idy=blockDim.y*blockIdx.y+threadIdx.y;
	if(idx<n && idy<n){
		int sum=0;
		for(int k=0;k<n;k++)
			sum+=a[idy*n+k]*b[k*n+idx];
		c[idy*n+idx]+=sum;
	}
}

__host__ int main(int argc,char* argv[]){
	srand((unsigned)time(NULL));
	struct timespec tstart={0,0}, tend={0,0};
	int block_size;
	const int n=1000;
	printf("Matrix size %d*%d\n",n,n);
	size_t bytes=sizeof(int)*n*n;
	int* a = (int*)malloc(bytes);
	int* b = (int*)malloc(bytes);
	int* c0 = (int*)malloc(bytes);
	int* c1 = (int*)malloc(bytes);
	int* c2 = (int*)malloc(bytes);
	int* d0 = (int*)malloc(bytes);
	int* cpu = (int*)malloc(bytes);
	init(a,n);
	init(b,n);
	memset(c0,0,bytes);
	memset(c1,0,bytes);
	memset(c2,0,bytes);
	memset(d0,0,bytes);
	memset(cpu,0,bytes);

	int* da=NULL;//device a
	int* db=NULL;
	int* dc=NULL;

	cuAssert(hipMalloc((void**)&da,bytes));
	cuAssert(hipMalloc((void**)&db,bytes));
	cuAssert(hipMalloc((void**)&dc,bytes));

	cuAssert(hipMemcpy(da,a,bytes,hipMemcpyHostToDevice));
	cuAssert(hipMemcpy(db,b,bytes,hipMemcpyHostToDevice));
	memsetGPU(dc,n);
	

//naive - non coalesed
	dim3 grid(n,n,1);
	dim3 block(1,1,1);
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	naive<<<grid,block>>>(da,db,dc,n);
	cuAssert(hipDeviceSynchronize());
    clock_gettime(CLOCK_MONOTONIC, &tend);
	printTime("GPU 01x01/v1",tstart,tend);
	cuAssert(hipGetLastError());
	cuAssert(hipMemcpy(c0,dc,bytes,hipMemcpyDeviceToHost));
	memsetGPU(dc,n);


//change block size to 32x32 66,6% usage
	block_size=32;
	block.x=block_size;
	block.y=block_size;
	grid.x=(n+block_size-1)/block_size;
	grid.y=(n+block_size-1)/block_size;
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	naive<<<grid,block>>>(da,db,dc,n);
	cuAssert(hipDeviceSynchronize());
    clock_gettime(CLOCK_MONOTONIC, &tend);
	printTime("GPU 32x32/v1",tstart,tend);
	cuAssert(hipMemcpy(c1,dc,bytes,hipMemcpyDeviceToHost));
	memsetGPU(dc,n);


//change block size to 16x16 100% usage
	block_size=16;
	block.x=block_size;
	block.y=block_size;
	grid.x=(n+block_size-1)/block_size;
	grid.y=(n+block_size-1)/block_size;
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	naive<<<grid,block>>>(da,db,dc,n);
	cuAssert(hipDeviceSynchronize());
    clock_gettime(CLOCK_MONOTONIC, &tend);
	printTime("GPU 16x16/v1",tstart,tend);
	cuAssert(hipMemcpy(c2,dc,bytes,hipMemcpyDeviceToHost));
	memsetGPU(dc,n);


//common algoritm block size to 16x16 100% usage
	block_size=16;
	block.x=block_size;
	block.y=block_size;
	grid.x=(n+block_size-1)/block_size;
	grid.y=(n+block_size-1)/block_size;
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	common<<<grid,block>>>(da,db,dc,n);
	cuAssert(hipDeviceSynchronize());
    clock_gettime(CLOCK_MONOTONIC, &tend);
	printTime("GPU 16x16/v2",tstart,tend);
	cuAssert(hipMemcpy(d0,dc,bytes,hipMemcpyDeviceToHost));
	memsetGPU(dc,n);

	cuAssert(hipFree(da));
	cuAssert(hipFree(db));
	cuAssert(hipFree(dc));

	
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	matrixMult(a,b,cpu,n);
    clock_gettime(CLOCK_MONOTONIC, &tend);
	printTime("CPU         ",tstart,tend);
	//printMatrix(cpu,n);
	//printMatrix(c0,n);
	//printMatrix(c1,n);
	printf("validate 01x01/v1 / cpu = %s\n", validate(c0,cpu,n) ? "true" : "false");
	printf("validate 32x32/v1 / cpu = %s\n", validate(c1,cpu,n) ? "true" : "false");
	printf("validate 16x16/v1 / cpu = %s\n", validate(c2,cpu,n) ? "true" : "false");
	printf("validate 16x16/v2 / cpu = %s\n", validate(d0,cpu,n) ? "true" : "false");

	free(a);
	free(b);
	free(c0);
	free(c1);
	free(c2);
	return 0;
}
